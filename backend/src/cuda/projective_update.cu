#include "hip/hip_runtime.h"
#include "projective_update.hpp"

#include "../common/field_impls.hpp"

#include <supereight/algorithms/filter.hpp>
#include <supereight/backend/cuda_util.hpp>
#include <supereight/functors/data_handler.hpp>
#include <supereight/functors/data_handler_cuda.hpp>

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

namespace se {

/*
template<typename OctreeT>
__global__ static void buildActiveList(
    OctreeT octree, OctreeT::block_type* active_list, int* idx) {}
*/

template<typename OctreeT>
__global__ static void updateBlockActiveKernel(OctreeT octree, Sophus::SE3f Tcw,
    Eigen::Matrix4f K, Eigen::Vector2i frame_size, int max_idx) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= max_idx) return;

    auto* block = octree.getBlockBuffer()[idx];
    if (!block->active()) {
        block->active(algorithms::in_frustum<OctreeT::block_type>(
            block, octree.voxel_size(), K * Tcw.matrix(), frame_size));
    }
}

template<typename OctreeT, typename UpdateFuncT>
__global__ static void __launch_bounds__(64)
    updateBlocksKernel(OctreeT octree, UpdateFuncT func, Sophus::SE3f Tcw,
        Eigen::Matrix4f K, Eigen::Vector2i frame_size) {
    auto* block = octree.getBlockBuffer()[blockIdx.x];

    if (!block->active() /* &&
        !algorithms::in_frustum<OctreeT::block_type>(
            block, octree.voxel_size(), K * Tcw.matrix(), frame_size)*/)
        return;

    typedef hipcub::BlockReduce<int, 64> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    const Eigen::Vector3i block_coord = block->coordinates();
    const Eigen::Vector3f pos_delta =
        Tcw.rotationMatrix() * Eigen::Vector3f(octree.voxel_size(), 0, 0);
    const Eigen::Vector3f camera_delta = K.topLeftCorner<3, 3>() * pos_delta;

    int num_visible = 0;

    int y = threadIdx.x + block_coord(1);
    int z = threadIdx.y + block_coord(2);

    Eigen::Vector3i pix   = Eigen::Vector3i(block_coord(0), y, z);
    Eigen::Vector3f start = Tcw *
        Eigen::Vector3f(block_coord(0) * octree.voxel_size(),
            y * octree.voxel_size(), z * octree.voxel_size());
    Eigen::Vector3f camera_start = K.topLeftCorner<3, 3>() * start;

    for (int x = 0; x < BLOCK_SIDE; ++x) {
        pix(0)                             = x + block_coord(0);
        const Eigen::Vector3f camera_voxel = camera_start + (x * camera_delta);
        const Eigen::Vector3f pos          = start + (x * pos_delta);
        if (pos(2) < 0.0001f) continue;

        const float inverse_depth = 1.f / camera_voxel(2);
        const Eigen::Vector2f pixel =
            Eigen::Vector2f(camera_voxel(0) * inverse_depth + 0.5f,
                camera_voxel(1) * inverse_depth + 0.5f);

        if (pixel(0) < 0.5f || pixel(0) > frame_size(0) - 1.5f ||
            pixel(1) < 0.5f || pixel(1) > frame_size(1) - 1.5f)
            continue;

        num_visible++;

        VoxelBlockHandlerCUDA<typename OctreeT::value_type> handler = {
            block, pix};
        func(handler, pix, pos, pixel);
    }

    int num_active = BlockReduce(temp_storage).Sum(num_visible, 64);
    if (threadIdx.x == 0 && threadIdx.y == 0) { block->active(num_active > 0); }
}

/*
template<typename OctreeT, typename UpdateFuncT>
__global__ static void updateBlocksKernel(OctreeT octree, UpdateFuncT func,
    Sophus::SE3f Tcw, Eigen::Matrix4f K, Eigen::Vector2i frame_size, int maxIdx,
    float* splat) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= maxIdx) return;

    auto block_buffer = octree.getBlockBuffer();
    auto* block       = block_buffer[idx];

    float voxel_size = octree.dim() / octree.size();

    if (!block->active() &&
        !algorithms::in_frustum<OctreeT::block_type>(
            block, voxel_size, K * Tcw.matrix(), frame_size))
        return;

    const Eigen::Vector3i blockCoord = block->coordinates();
    const Eigen::Vector3f delta =
        Tcw.rotationMatrix() * Eigen::Vector3f(voxel_size, 0, 0);
    const Eigen::Vector3f cameraDelta = K.topLeftCorner<3, 3>() * delta;
    bool is_visible                   = false;

    unsigned int y, z;
    unsigned int ylast = blockCoord(1) + BLOCK_SIDE;
    unsigned int zlast = blockCoord(2) + BLOCK_SIDE;

    for (z = blockCoord(2); z < zlast; ++z) {
        for (y = blockCoord(1); y < ylast; ++y) {
            Eigen::Vector3i pix   = Eigen::Vector3i(blockCoord(0), y, z);
            Eigen::Vector3f start = Tcw *
                Eigen::Vector3f((pix(0)) * voxel_size, (pix(1)) * voxel_size,
                    (pix(2)) * voxel_size);
            Eigen::Vector3f camerastart = K.topLeftCorner<3, 3>() * start;
            for (unsigned int x = 0; x < BLOCK_SIDE; ++x) {
                pix(0) = x + blockCoord(0);
                const Eigen::Vector3f camera_voxel =
                    camerastart + (x * cameraDelta);
                const Eigen::Vector3f pos = start + (x * delta);
                if (pos(2) < 0.0001f) continue;

                const float inverse_depth = 1.f / camera_voxel(2);
                const Eigen::Vector2f pixel =
                    Eigen::Vector2f(camera_voxel(0) * inverse_depth + 0.5f,
                        camera_voxel(1) * inverse_depth + 0.5f);
                if (pixel(0) < 0.5f || pixel(0) > frame_size(0) - 1.5f ||
                    pixel(1) < 0.5f || pixel(1) > frame_size(1) - 1.5f)
                    continue;
                is_visible = true;

                const auto pixel_loc = pixel.cast<int>();
                float* splat_px =
                    &splat[pixel_loc.x() + frame_size.x() * pixel_loc.y()];

                VoxelBlockHandler<typename OctreeT::value_type> handler = {
                    block, pix};
                bool near = func(handler, pix, pos, pixel);

                if (near) {
                    float dist = camera_voxel(2);
                    atomicMin(
                        reinterpret_cast<int*>(splat_px), __float_as_int(dist));
                }
            }
        }
    }

    block->active(is_visible);
}
*/

template<typename OctreeT, typename UpdateFuncT>
__global__ static void updateNodesKernel(OctreeT octree, UpdateFuncT func,
    Sophus::SE3f Tcw, Eigen::Matrix4f K, Eigen::Vector2i frame_size,
    int maxIdx) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= maxIdx) return;

    auto node_buffer = octree.getNodesBuffer();
    auto* node       = node_buffer[idx];

    const Eigen::Vector3i voxel = Eigen::Vector3i(unpack_morton(node->code_));
    const Eigen::Vector3f delta = Tcw.rotationMatrix() *
        Eigen::Vector3f::Constant(0.5f * octree.voxel_size() * node->side_);

    const Eigen::Vector3f delta_c = K.topLeftCorner<3, 3>() * delta;

    Eigen::Vector3f base_cam =
        Tcw * (octree.voxel_size() * voxel.cast<float>());
    Eigen::Vector3f basepix_hom = K.topLeftCorner<3, 3>() * base_cam;

    for (int i = 0; i < 8; ++i) {
        const Eigen::Vector3i dir =
            Eigen::Vector3i((i & 1) > 0, (i & 2) > 0, (i & 4) > 0);
        const Eigen::Vector3f vox_cam =
            base_cam + dir.cast<float>().cwiseProduct(delta);
        const Eigen::Vector3f pix_hom =
            basepix_hom + dir.cast<float>().cwiseProduct(delta_c);

        if (vox_cam(2) < 0.0001f) continue;
        const float inverse_depth = 1.f / pix_hom(2);
        const Eigen::Vector2f pixel =
            Eigen::Vector2f(pix_hom(0) * inverse_depth + 0.5f,
                pix_hom(1) * inverse_depth + 0.5f);
        if (pixel(0) < 0.5f || pixel(0) > frame_size(0) - 1.5f ||
            pixel(1) < 0.5f || pixel(1) > frame_size(1) - 1.5f)
            continue;

        NodeHandler<FieldType> handler = {node, i};
        func(handler, voxel + dir, vox_cam, pixel);
    }
}

static void updateBlocks(Octree<FieldType, MemoryPoolCUDA>& octree,
    voxel_traits<FieldType>::update_func_type& func, Sophus::SE3f Tcw,
    Eigen::Matrix4f K, Eigen::Vector2i frame_size) {
    auto& block_buffer = octree.getBlockBuffer();
    int num_elem       = block_buffer.used();
    if (num_elem < 1) return;

    dim3 threads(BLOCK_SIDE, BLOCK_SIDE);
    dim3 blocks(num_elem);

    updateBlockActiveKernel<<<(num_elem + 255) / 256, 256>>>(
        octree, Tcw, K, frame_size, num_elem);

    updateBlocksKernel<<<blocks, threads>>>(octree, func, Tcw, K, frame_size);
    safeCall(hipPeekAtLastError());
}

static void updateNodes(Octree<FieldType, MemoryPoolCUDA>& octree,
    voxel_traits<FieldType>::update_func_type& func, Sophus::SE3f Tcw,
    Eigen::Matrix4f K, Eigen::Vector2i frame_size) {
    auto& node_buffer = octree.getNodesBuffer();
    int num_elem      = node_buffer.used();

    if (num_elem < 1) return;

    updateNodesKernel<<<(num_elem + 255) / 256, 256>>>(
        octree, func, Tcw, K, frame_size, num_elem);
    safeCall(hipPeekAtLastError());
}

void projectiveUpdate(Octree<FieldType, MemoryPoolCUDA>& octree,
    voxel_traits<FieldType>::update_func_type& func, Sophus::SE3f Tcw,
    Eigen::Matrix4f K, Eigen::Vector2i frame_size) {
    updateBlocks(octree, func, Tcw, K, frame_size);
    updateNodes(octree, func, Tcw, K, frame_size);
}

} // namespace se
